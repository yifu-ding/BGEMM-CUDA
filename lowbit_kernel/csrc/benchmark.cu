#include "hip/hip_runtime.h"
#include "fp6_linear.cu"
#include "bgemm.cu"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "kernel_test.h"

#define SAVE_IO
#define BENCHMARK_MODE

// bgemm
int main(int argc, char** argv)
{
    // Parsing the inputs from CLI.
    // int dev = findCudaDevice(argc, (const char **)argv);
    // printf(dev);
    if (argc != 5) {
        printf("Wrong Inputs! Correct input format: ./main #Row_Weight #Column_Weight BatchSize SplitK\n");
        return -1;
    }
    size_t M_GLOBAL = atoi(argv[1]);
    size_t K_GLOBAL = atoi(argv[2]);
    size_t N_GLOBAL = atoi(argv[3]);
    int    SPLIT_K  = atoi(argv[4]);
    // assert(M_GLOBAL%256==0);                 // Currently, M_GLOBAL must be a multiple of 256.
    // assert(K_GLOBAL%64==0);                  // Currently, K_GLOBAL must be a multiple of 64.

    // Matrices in quantized Bin models with faked values.
    // unsigned char: 1 Byte = 0~255
    unsigned char* A_1bit_h  = (unsigned char*)malloc(M_GLOBAL*K_GLOBAL*1/8);       CheckMallocCPU(A_1bit_h, __LINE__);     // Weight matrix with FP6 values, stored in row-major.
    for(size_t i=0; i<M_GLOBAL*K_GLOBAL*1/8; i++)   A_1bit_h[i] = rand() % 256;    // noqa                                         // Random initialization.
    #ifdef SAVE_IO
        print_uint32((uint32_t*) A_1bit_h,  "A_1bit_h", M_GLOBAL, K_GLOBAL);
    #endif
    half*          A_Scale_h = (half*)malloc(M_GLOBAL*sizeof(half));                CheckMallocCPU(A_Scale_h, __LINE__);    // Quantization Scales with FP16 values.
    for(size_t i=0; i<M_GLOBAL; i++)                A_Scale_h[i] = 1.0f;// float(rand()%256)/64.0f;                                 // Scale
    // Generaing FP16 format of the Weight Matrix
    half* A_16bit_h = (half*) malloc(M_GLOBAL*K_GLOBAL*sizeof(half));                           CheckMallocCPU(A_16bit_h, __LINE__);
    DeQuantMatrix_B1_To_FP16(A_16bit_h, A_1bit_h, M_GLOBAL, K_GLOBAL, A_Scale_h);
    // In-place weight pre-packing
    // weight_matrix_prepacking((int*)A_1bit_h, (int*)A_1bit_h, M_GLOBAL, K_GLOBAL);  // noqa: no need?
    #ifdef SAVE_IO
        print_binary(A_16bit_h,  "A_16bit_h",  M_GLOBAL, K_GLOBAL);
    #endif
    // print_binary(A_16bit_h,  "A_16bit_h",  M_GLOBAL, K_GLOBAL);

    // Matrices in quantized Bin models with faked values.
    unsigned char* B_1bit_h  = (unsigned char*)malloc(N_GLOBAL*K_GLOBAL*1/8);       CheckMallocCPU(B_1bit_h, __LINE__);     // Weight matrix with FP6 values, stored in row-major.
    for(size_t i=0; i<N_GLOBAL*K_GLOBAL*1/8; i++)   B_1bit_h[i] = rand() % 256;    // noqa                                         // Random initialization.
    #ifdef SAVE_IO
        print_uint32((uint32_t*) B_1bit_h, "B_1bit_h", N_GLOBAL, K_GLOBAL);
    #endif
    half*          B_Scale_h = (half*)malloc(N_GLOBAL*sizeof(half));                CheckMallocCPU(B_Scale_h, __LINE__);    // Quantization Scales with FP16 values.
    for(size_t i=0; i<N_GLOBAL; i++)                B_Scale_h[i] = 1.0; // float(rand()%256)/64.0f;                                 // Scale
    // Generaing FP16 format of the Weight Matrix
    half* B_16bit_h = (half*) malloc(N_GLOBAL*K_GLOBAL*sizeof(half));                           CheckMallocCPU(B_16bit_h, __LINE__);
    DeQuantMatrix_B1_To_FP16(B_16bit_h, B_1bit_h, N_GLOBAL, K_GLOBAL, B_Scale_h);
    #ifdef SAVE_IO
        print_binary(B_16bit_h, "B_16bit_h", N_GLOBAL, K_GLOBAL);
    #endif
    // Devices Memory
    unsigned char*  A_1bit;  // 1B = 8b
    half*           A_Scale;  // 16b
    half*           A_16bit;  // 16b
    hipMalloc(reinterpret_cast<void**>(&A_1bit),  M_GLOBAL*K_GLOBAL*1/8);             CheckMallocCUDA(A_1bit, __LINE__);
    hipMalloc(reinterpret_cast<void**>(&A_Scale), M_GLOBAL*sizeof(half));             CheckMallocCUDA(A_Scale, __LINE__);
    hipMalloc(reinterpret_cast<void**>(&A_16bit),          M_GLOBAL*K_GLOBAL*sizeof(half));    CheckMallocCUDA(A_16bit, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(A_1bit,     A_1bit_h,  M_GLOBAL*K_GLOBAL*1/8,          hipMemcpyHostToDevice);
    hipMemcpy(A_Scale,    A_Scale_h,          M_GLOBAL*sizeof(half),          hipMemcpyHostToDevice);
    hipMemcpy(A_16bit,             A_16bit_h,          M_GLOBAL*K_GLOBAL*sizeof(half), hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
    // Devices Memory
    unsigned char*  B_1bit;  // 1B = 8b
    half*           B_Scale;  // 16b
    half*           B_16bit;  // 16b
    hipMalloc(reinterpret_cast<void**>(&B_1bit),  N_GLOBAL*K_GLOBAL*1/8);             CheckMallocCUDA(B_1bit, __LINE__);
    hipMalloc(reinterpret_cast<void**>(&B_Scale), N_GLOBAL*sizeof(half));             CheckMallocCUDA(B_Scale, __LINE__);
    hipMalloc(reinterpret_cast<void**>(&B_16bit),          N_GLOBAL*K_GLOBAL*sizeof(half));    CheckMallocCUDA(B_16bit, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(B_1bit,     B_1bit_h,  N_GLOBAL*K_GLOBAL*1/8,          hipMemcpyHostToDevice);
    hipMemcpy(B_Scale,    B_Scale_h,          N_GLOBAL*sizeof(half),          hipMemcpyHostToDevice);
    hipMemcpy(B_16bit,             B_16bit_h,          N_GLOBAL*K_GLOBAL*sizeof(half), hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // B Matrix: Activations
    /* half* B_h = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL); CheckMallocCPU(B_h);       // col major 
    for (size_t i = 0; i < N_GLOBAL * K_GLOBAL; i++)
        B_h[i] = __float2half_rn(static_cast<float>((rand() % 5)) / 5 - 0.5f);
    // Device memory
    half* B            = NULL;
    hipMalloc(reinterpret_cast<void**>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);               CheckMallocCUDA(B, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice); */
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipblasStatus_t cublas_status;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    half* D_cublas = NULL;
    #ifdef BENCHMARK_MODE
    printf("Launching CuBlas...\n");
    hipMalloc(reinterpret_cast<void**>(&D_cublas), sizeof(half) * M_GLOBAL * N_GLOBAL);        CheckMallocCUDA(D_cublas, __LINE__);
    hipMemset(D_cublas, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);
    //hipblasSetMathMode(handle, HIPBLAS_PEDANTIC_MATH);          // Tensor core NOT enabled
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);             // Tensor core enabled
    hipDeviceSynchronize(); 
    int              m = M_GLOBAL, n = N_GLOBAL, k = K_GLOBAL;
    const float      alpha     = 1.0;
    const float      beta      = 0.0;
    hipblasGemmAlgo_t CuBlasALG = static_cast<hipblasGemmAlgo_t>(0);
    for (int i = 0; i < WARM_UP_ITERATION; i++) {
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,   HIPBLAS_OP_N,
                                     m, n, k,
                                     &alpha,
                                     A_16bit,   HIP_R_16F, k,
                                     B_16bit,   HIP_R_16F, k,
                                     &beta,
                                     D_cublas,  HIP_R_16F, m,
                                     HIP_R_32F,
                                     CuBlasALG);
        checkCublasError(cublas_status, __LINE__);
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,   HIPBLAS_OP_N,
                                     m, n, k,
                                     &alpha,
                                     A_16bit,   HIP_R_16F, k,
                                     B_16bit,   HIP_R_16F, k,
                                     &beta,
                                     D_cublas,  HIP_R_16F, m,
                                     HIP_R_32F,
                                     CuBlasALG);
    hipEventRecord(stop); 
    hipEventSynchronize(stop);
    #endif
    float milliseconds_cublas = 0;
    hipEventElapsedTime(&milliseconds_cublas, start, stop); 
    milliseconds_cublas = milliseconds_cublas / BENCHMARK_ITERATION;
    float tflops_cublas = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_cublas / 1000.)) / 1e12;
    //
    
    half* D_cublas_h = NULL;  // col major
    #ifdef BENCHMARK_MODE
    D_cublas_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);   CheckMallocCPU(D_cublas_h);
    hipMemcpy(D_cublas_h, D_cublas, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_cublas);  
    #endif
    // print_half(D_cublas_h, "D_cublas_h", M_GLOBAL, N_GLOBAL);

    // checkLastCudaError(__LINE__);
    printf("Launching BGEMM...\n");
    half* D_bin = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_bin), sizeof(half) * M_GLOBAL * N_GLOBAL); CheckMallocCUDA(D_bin);
    hipMemset(D_bin, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    
    int Split_K = SPLIT_K;
    float* Reduction_Workspace = NULL;
    hipMalloc(reinterpret_cast<void**>(&Reduction_Workspace), sizeof(float) * M_GLOBAL * N_GLOBAL * Split_K);   CheckMallocCUDA(Reduction_Workspace, __LINE__);
    //

    for (int i = 0; i < WARM_UP_ITERATION; i++)
        bin_linear_kernel(  0,
                        (uint4*)A_1bit, A_Scale, B_Scale,
                        (uint32_t*) B_1bit,
                        D_bin,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        Split_K);
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        bin_linear_kernel(  0,
                        (uint4*) A_1bit, A_Scale, B_Scale,
                        (uint32_t*) B_1bit,
                        D_bin,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        Split_K);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    checkLastCudaError(__LINE__);
    //
    float milliseconds_bin = 0.0f;
    hipEventElapsedTime(&milliseconds_bin, start, stop);
    milliseconds_bin = milliseconds_bin / BENCHMARK_ITERATION;
    float tflops_bin = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_bin / 1000.)) / 1e12;
    half* D_bin_h = NULL;  // col major
    D_bin_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipMemcpy(D_bin_h, D_bin, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_bin);
    hipFree(Reduction_Workspace);
    /////////////////////////////////////////////////////////////////////////////////////////////////
    
    printf("Verifying correctness of the computations...\n");
    double totalRelativeError_bin  = ComputeTotalError(D_cublas_h, D_bin_h, M_GLOBAL, N_GLOBAL);

    // #ifdef BENCHMARK_MODE
    // /////////////////////////////////////////////////////////////////////////////////////////////////
    // printf("Launching FP6-LLM...\n");
    // half* D_fp6 = NULL;
    // hipMalloc(reinterpret_cast<void**>(&D_fp6), sizeof(half) * M_GLOBAL * N_GLOBAL); CheckMallocCUDA(D_fp6);
    // hipMemset(D_fp6, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    // //
    // for (int i = 0; i < WARM_UP_ITERATION; i++)
    //     fp6_linear_kernel(  0,
    //                     (uint4*)A_1bit, A_Scale,
    //                     B_16bit,
    //                     D_fp6,
    //                     M_GLOBAL, N_GLOBAL, K_GLOBAL,
    //                     Reduction_Workspace,  
    //                     Split_K);
    // hipEventRecord(start);
    // for (int i = 0; i < BENCHMARK_ITERATION; i++)
    //     fp6_linear_kernel(  0,
    //                     (uint4*)A_1bit, A_Scale,
    //                     B_16bit,
    //                     D_fp6,
    //                     M_GLOBAL, N_GLOBAL, K_GLOBAL,
    //                     Reduction_Workspace,  
    //                     Split_K);
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // checkLastCudaError(__LINE__);
    // //
    // float milliseconds_fp6 = 0.0f;
    // hipEventElapsedTime(&milliseconds_fp6, start, stop);
    // milliseconds_fp6 = milliseconds_fp6 / BENCHMARK_ITERATION;
    // float tflops_fp6 = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_fp6 / 1000.)) / 1e12;
    // half* D_fp6_h = NULL;  // col major
    // D_fp6_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    // hipMemcpy(D_fp6_h, D_fp6, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    // hipFree(D_fp6);
    // hipFree(Reduction_Workspace);
    // /////////////////////////////////////////////////////////////////////////////////////////////////
    // printf("Verifying correctness of the computations...\n");
    // double totalRelativeError_fp6  = ComputeTotalError(D_cublas_h, D_fp6_h, M_GLOBAL, N_GLOBAL);
    // #endif

    printf("M: %d N: %d K: %d SplitK: %d Iter: %d\n", M_GLOBAL, N_GLOBAL, K_GLOBAL, SPLIT_K, BENCHMARK_ITERATION);
    printf("******************************************Performance*******************************************\n");
    PrintPerformance("cuBLAS", milliseconds_cublas, tflops_cublas, 0.0);
    // #ifdef BENCHMARK_MODE
    // PrintPerformance("fp6_llm", milliseconds_fp6, tflops_fp6, totalRelativeError_fp6);
    // #endif
    PrintPerformance("BGEMM", milliseconds_bin, tflops_bin, totalRelativeError_bin);
    #ifdef SAVE_IO
        PrintResult("BGEMM", 100, D_cublas_h, D_bin_h, M_GLOBAL, N_GLOBAL);
    #endif
    #ifdef DEBUG_MODE
        PrintMismatch("BGEMM", 100, 0.000, D_cublas_h, D_bin_h, M_GLOBAL, N_GLOBAL);
    #endif

    free(D_cublas_h);
    free(D_bin_h);
    return 0;
}