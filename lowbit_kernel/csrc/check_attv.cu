#include "hip/hip_runtime.h"
#include "fp6_linear.cu"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <omp.h>			//声明引用OpenMP库

#define NUM_THREADS 128

// #include "kernel_test.h"

// #define SAVE_IO
#define BENCHMARK_MODE
// #define DEBUG_MODE

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int getThreadNum()
{
    hipDeviceProp_t prop;
    int count;
    
    // HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK = 128;

    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n", count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: %d, %d, %d)\n",
     prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    // printf("%d\n", HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK);
    return prop.maxThreadsPerBlock;
}


// bgemm
int main(int argc, char** argv)
{

    // Parsing the inputs from CLI.
    // int dev = findCudaDevice(argc, (const char **)argv);
    // printf(dev);
    if (argc != 5) {
        printf("Wrong Inputs! Correct input format: ./main #Row_Weight #Column_Weight BatchSize SplitK\n");
        return -1;
    }
    size_t M_GLOBAL = atoi(argv[1]);
    size_t K_GLOBAL = atoi(argv[2]);
    size_t N_GLOBAL = atoi(argv[3]);
    int    SPLIT_K  = atoi(argv[4]);

    getThreadNum();

    // uint32_t* packed_weights = (uint32_t*) malloc(M_GLOBAL*K_GLOBAL*sizeof(uint32_t)/32);
    // half* ori_weight = (half*) malloc(M_GLOBAL*K_GLOBAL*sizeof(half));
    // for(size_t i=0; i<M_GLOBAL*K_GLOBAL; i++)   ori_weight[i] = (rand() % 100 - 50) / 17.0f; 
   
    // bin_matrix_prepacking_to_uint32(packed_weights, ori_weight, M_GLOBAL, K_GLOBAL);
    // #ifdef SAVE_IO
    //     print_half(ori_weight, "ori_weight", M_GLOBAL, K_GLOBAL);
    //     print_uint32(packed_weights, "packed_weights", M_GLOBAL, K_GLOBAL);
    // #endif

    // return 0;

    // #ifndef BENCHMARK_MODE
    //     auto weight = torch::randn({(signed long) M_GLOBAL, (signed long) K_GLOBAL}, torch::kFloat16);//.to(torch::kCUDA);
    //     auto feat = torch::randn({(signed long) N_GLOBAL, (signed long) K_GLOBAL}, torch::kFloat16);//.to(torch::kCUDA);
    //     auto scale = torch::randn({(signed long) M_GLOBAL}, torch::kFloat16);//.to(torch::kCUDA);

    //     auto res = bgemm_linear_forward_cuda(feat, weight, scale, 1);

    //     return 0;
    // #endif


    // assert(M_GLOBAL%256==0);                 // Currently, M_GLOBAL must be a multiple of 256.
    // assert(K_GLOBAL%64==0);                  // Currently, K_GLOBAL must be a multiple of 64.



    //////////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////////
    // Matrices in quantized Bin models with faked values.
    // unsigned char: 1 Byte = 0~255
    // unsigned char* A_1bit_h  = (unsigned char*)malloc(M_GLOBAL*K_GLOBAL*1/8);       CheckMallocCPU(A_1bit_h, __LINE__);     // Weight matrix with FP6 values, stored in row-major.
    // for(size_t i=0; i<M_GLOBAL*K_GLOBAL*1/8; i++)   A_1bit_h[i] = rand() % 256;    // noqa                                         // Random initialization.

    // half*          A_Scale_h = (half*)malloc(M_GLOBAL*sizeof(half));                CheckMallocCPU(A_Scale_h, __LINE__);    // Quantization Scales with FP16 values.
    // for(size_t i=0; i<M_GLOBAL; i++)                A_Scale_h[i] = 1.0f;// float(rand()%256)/64.0f;                                 // Scale
    // // Generaing FP16 format of the Weight Matrix
    // half* A_16bit_h = (half*) malloc(M_GLOBAL*K_GLOBAL*sizeof(half));                           CheckMallocCPU(A_16bit_h, __LINE__);
    // DeQuantMatrix_B1_To_FP16(A_16bit_h, A_1bit_h, M_GLOBAL, K_GLOBAL, A_Scale_h);
    // // In-place weight pre-packing
    // // weight_matrix_prepacking((int*)A_1bit_h, (int*)A_1bit_h, M_GLOBAL, K_GLOBAL);  // noqa: no need?
    // #ifdef SAVE_IO
    //     print_binary(A_16bit_h,  "A_16bit_h",  M_GLOBAL, K_GLOBAL);
    // #endif

    // // Matrices in quantized Bin models with faked values.
    // unsigned char* B_1bit_h  = (unsigned char*)malloc(N_GLOBAL*K_GLOBAL*1/8);       CheckMallocCPU(B_1bit_h, __LINE__);     // Weight matrix with FP6 values, stored in row-major.
    // for(size_t i=0; i<N_GLOBAL*K_GLOBAL*1/8; i++)   B_1bit_h[i] = rand() % 256;    // noqa                                         // Random initialization.
    // #ifdef SAVE_IO
    //     print_uint32((uint32_t*) B_1bit_h, "B_1bit_h", N_GLOBAL, K_GLOBAL);
    // #endif
    // half*          B_Scale_h = (half*)malloc(N_GLOBAL*sizeof(half));                CheckMallocCPU(B_Scale_h, __LINE__);    // Quantization Scales with FP16 values.
    // for(size_t i=0; i<N_GLOBAL; i++)                B_Scale_h[i] = 1.0; // float(rand()%256)/64.0f;                                 // Scale
    // // Generaing FP16 format of the Weight Matrix
    // // half* B_16bit_h = (half*) malloc(N_GLOBAL*K_GLOBAL*sizeof(half));                           CheckMallocCPU(B_16bit_h, __LINE__);
    // // DeQuantMatrix_B1_To_FP16(B_16bit_h, B_1bit_h, N_GLOBAL, K_GLOBAL, B_Scale_h);
    // // #ifdef SAVE_IO
    // //     print_binary(B_16bit_h, "B_16bit_h", N_GLOBAL, K_GLOBAL);
    // // #endif
    // // Devices Memory
    // uint32_t *  A_1bit;  // 1B = 8b
    // unsigned char *  A_1bit_convert = (unsigned char*)malloc(M_GLOBAL*K_GLOBAL*1/8);       CheckMallocCPU(A_1bit_convert, __LINE__); ;  // 1B = 8b
    // half*           A_Scale;  // 16b
    // half*           A_16bit;  // 16b
    // hipMalloc(reinterpret_cast<void**>(&A_1bit),  M_GLOBAL*K_GLOBAL*1/8);             CheckMallocCUDA(A_1bit, __LINE__);
    // hipMalloc(reinterpret_cast<void**>(&A_Scale), M_GLOBAL*sizeof(half));             CheckMallocCUDA(A_Scale, __LINE__);
    // hipMalloc(reinterpret_cast<void**>(&A_16bit),          M_GLOBAL*K_GLOBAL*sizeof(half));    CheckMallocCUDA(A_16bit, __LINE__);
    // // Memory Copy from CPU to GPU
    // convert_uchar2uint32_order(A_1bit_convert, A_1bit_h, M_GLOBAL, K_GLOBAL);
    // #ifdef SAVE_IO
    //     print_uint32((uint32_t*) A_1bit_convert,  "A_1bit_h",  M_GLOBAL, K_GLOBAL);
    // #endif
    // hipMemcpy(A_1bit,     A_1bit_convert,  M_GLOBAL*K_GLOBAL*1/8,          hipMemcpyHostToDevice);
    // hipMemcpy(A_Scale,    A_Scale_h,          M_GLOBAL*sizeof(half),          hipMemcpyHostToDevice);
    // hipMemcpy(A_16bit,             A_16bit_h,          M_GLOBAL*K_GLOBAL*sizeof(half), hipMemcpyHostToDevice);
    // checkLastCudaError(__LINE__);

    //////////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////////

    
    // Devices Memory
    // unsigned char*  B_1bit;  // 1B = 8b
    // half*           B_Scale;  // 16b
    // half*           B_16bit;  // 16b
    // hipMalloc(reinterpret_cast<void**>(&B_1bit),  N_GLOBAL*K_GLOBAL*1/8);             CheckMallocCUDA(B_1bit, __LINE__);
    // hipMalloc(reinterpret_cast<void**>(&B_Scale), N_GLOBAL*sizeof(half));             CheckMallocCUDA(B_Scale, __LINE__);
    // hipMalloc(reinterpret_cast<void**>(&B_16bit),          N_GLOBAL*K_GLOBAL*sizeof(half));    CheckMallocCUDA(B_16bit, __LINE__);
    // // Memory Copy from CPU to GPU
    // hipMemcpy(B_1bit,     B_1bit_h,  N_GLOBAL*K_GLOBAL*1/8,          hipMemcpyHostToDevice);
    // hipMemcpy(B_Scale,    B_Scale_h,          N_GLOBAL*sizeof(half),          hipMemcpyHostToDevice);
    // hipMemcpy(B_16bit,             B_16bit_h,          N_GLOBAL*K_GLOBAL*sizeof(half), hipMemcpyHostToDevice);
    // checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // fp16 matrix initialization for weight and activation
    
    // matrix A
    half* A_h = (half*)malloc(sizeof(half) *  M_GLOBAL * K_GLOBAL); CheckMallocCPU(A_h);       // col major 
    for (size_t i = 0; i < M_GLOBAL * K_GLOBAL; i++)
        A_h[i] = __float2half_rn(static_cast<float>((rand() % 256)) / 128.0f - 1.0f);
    #ifdef SAVE_IO
        print_half(A_h,  "A_fp16", M_GLOBAL, K_GLOBAL); 
    #endif
    // Device memory
    half* A            = NULL;
    hipMalloc(reinterpret_cast<void**>(&A), sizeof(half) * M_GLOBAL * K_GLOBAL);               CheckMallocCUDA(A, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);

    half* B_h = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL); CheckMallocCPU(B_h);       // col major 
    for (size_t i = 0; i < N_GLOBAL * K_GLOBAL; i++)
        B_h[i] = __float2half_rn(static_cast<float>((rand() % 256)) / 128.0f - 1.0f);
    #ifdef SAVE_IO
        print_half(B_h,  "B_fp16", N_GLOBAL, K_GLOBAL);
    #endif
    // Device memory
    half* B            = NULL;
    hipMalloc(reinterpret_cast<void**>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);               CheckMallocCUDA(B, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    half* A_h_cublas = (half*)malloc(sizeof(half) *  M_GLOBAL * K_GLOBAL);   CheckMallocCPU(A_h_cublas);  
    half* B_h_cublas = (half*)malloc(sizeof(half) *  K_GLOBAL * N_GLOBAL);   CheckMallocCPU(B_h_cublas);       // col major 

    // matrix A
    for (size_t i = 0; i < M_GLOBAL * K_GLOBAL; i++) {
        if (__half2float(A_h[i]) > 0.0f) {
            A_h_cublas[i] = __float2half_rn(-1.0f);
        } else {
            A_h_cublas[i] = __float2half_rn(1.0f);
        }
    }
    // matrix B
    for (size_t i = 0; i < N_GLOBAL * K_GLOBAL; i++) {
        if (__half2float(B_h[i]) < -0.5f) {
            B_h_cublas[i] = __float2half_rn(-1.0f);
        } else if (__half2float(B_h[i]) > 0.5f) {
            B_h_cublas[i] =  __float2half_rn(1.0f);
        } else {
            B_h_cublas[i] =  __float2half_rn(0.0f);
        }
    } 
    
    // Device memory
    half* A_cublas            = NULL;
    hipMalloc(reinterpret_cast<void**>(&A_cublas), sizeof(half) * M_GLOBAL * K_GLOBAL);     CheckMallocCUDA(A_cublas, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(A_cublas, A_h_cublas, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);

    half* B_cublas            = NULL;
    hipMalloc(reinterpret_cast<void**>(&B_cublas), sizeof(half) * N_GLOBAL * K_GLOBAL);     CheckMallocCUDA(B_cublas, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(B_cublas, B_h_cublas, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);


    hipblasStatus_t cublas_status;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    half* D_cublas = NULL;
    #ifdef BENCHMARK_MODE
    printf("Launching CuBlas...\n");
    hipMalloc(reinterpret_cast<void**>(&D_cublas), sizeof(half) * M_GLOBAL * N_GLOBAL);        CheckMallocCUDA(D_cublas, __LINE__);
    hipMemset(D_cublas, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);
   
    //hipblasSetMathMode(handle, HIPBLAS_PEDANTIC_MATH);          // Tensor core NOT enabled
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);             // Tensor core enabled
    hipDeviceSynchronize(); 
    int              m = M_GLOBAL, n = N_GLOBAL, k = K_GLOBAL;
    const float      alpha     = 1.0;
    const float      beta      = 0.0;
    hipblasGemmAlgo_t CuBlasALG = static_cast<hipblasGemmAlgo_t>(0);
    // hipDeviceSetLimit();
    
    for (int i = 0; i < WARM_UP_ITERATION; i++) {
        // #pragma omp parallel num_threads(NUM_THREADS)
        // {
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,   HIPBLAS_OP_N,
                                     m, n, k,
                                     &alpha,
                                     A_cublas,   HIP_R_16F, k,
                                     B_cublas,   HIP_R_16F, k,
                                     &beta,
                                     D_cublas,  HIP_R_16F, m,
                                     HIP_R_32F,
                                     CuBlasALG);
        checkCublasError(cublas_status, __LINE__);
        // printf("%d ", omp_get_thread_num());
        // }
    }
    printf("\n");
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        // #pragma omp parallel num_threads(NUM_THREADS)
        // {
        {    
        cublas_status = hipblasGemmEx(handle,
                                        HIPBLAS_OP_T,   HIPBLAS_OP_N,
                                        m, n, k,
                                        &alpha,
                                        A_cublas,   HIP_R_16F, k,
                                        B_cublas,   HIP_R_16F, k,
                                        &beta,
                                        D_cublas,  HIP_R_16F, m,
                                        HIP_R_32F,
                                        CuBlasALG);
        // printf("%d ", omp_get_thread_num());
        }
    
        // }
    hipEventRecord(stop); 
    hipEventSynchronize(stop);

    float milliseconds_cublas = 0;
    hipEventElapsedTime(&milliseconds_cublas, start, stop); 
    milliseconds_cublas = milliseconds_cublas / BENCHMARK_ITERATION;
    float tflops_cublas = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_cublas / 1000.)) / 1e12;
    //
    
    half* D_cublas_h = NULL;  // col major
    D_cublas_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);   CheckMallocCPU(D_cublas_h);
    hipMemcpy(D_cublas_h, D_cublas, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_cublas);  
    

    //////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////
    // checkLastCudaError(__LINE__);
    printf("Launching FP GEMM...\n");
    half* D_fp = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_fp), sizeof(half) * M_GLOBAL * N_GLOBAL); CheckMallocCUDA(D_fp);
    hipMemset(D_fp, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    
    float* Reduction_Workspace = NULL;
    hipMalloc(reinterpret_cast<void**>(&Reduction_Workspace), sizeof(float) * M_GLOBAL * N_GLOBAL * 1);   CheckMallocCUDA(Reduction_Workspace, __LINE__);
    
    for (int i = 0; i < WARM_UP_ITERATION; i++)
    {
        // #pragma omp parallel num_threads(NUM_THREADS)
        // {
        fp16_linear_kernel(  0,
                        A_cublas, // A_Scale, B_Scale,
                        B,
                        D_fp,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        1);
        // }
    }
    // }
    hipEventRecord(start);
    // #pragma omp parallel num_threads(NUM_THREADS)
    // {
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
    {   
        // #pragma omp parallel num_threads(NUM_THREADS)
        // {
        fp16_linear_kernel(  0,
                        A_cublas, // A_Scale, B_Scale,
                        B,
                        D_fp,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        1);
        // }
    }
    // }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    checkLastCudaError(__LINE__);
    //
    float milliseconds_fp = 0.0f;
    hipEventElapsedTime(&milliseconds_fp, start, stop);
    milliseconds_fp = milliseconds_fp / BENCHMARK_ITERATION;
    float tflops_fp = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_fp / 1000.)) / 1e12;
    half* D_fp_h = NULL;  // col major
    D_fp_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipMemcpy(D_fp_h, D_fp, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_fp);
    hipFree(Reduction_Workspace);

    //////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////
   
    #endif 
    
    // print_half(D_cublas_h, "D_cublas_h", M_GLOBAL, N_GLOBAL);

    // checkLastCudaError(__LINE__);
    printf("Launching W2A3 Att*V...\n");
    half* D_bin = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_bin), sizeof(half) * M_GLOBAL * N_GLOBAL); CheckMallocCUDA(D_bin);
    hipMemset(D_bin, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    
    int Split_K = SPLIT_K;
    Reduction_Workspace = NULL;
    hipMalloc(reinterpret_cast<void**>(&Reduction_Workspace), sizeof(float) * M_GLOBAL * N_GLOBAL * Split_K);   CheckMallocCUDA(Reduction_Workspace, __LINE__);
    //
    /* for (int i = 0; i < WARM_UP_ITERATION; i++)
        bin_linear_kernel(  0,
                        (uint4*)A_1bit, A_Scale, B_Scale,
                        (uint32_t*) B_1bit,
                        D_bin,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        Split_K);
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        bin_linear_kernel(  0,
                        (uint4*) A_1bit, A_Scale, B_Scale,
                        (uint32_t*) B_1bit,
                        D_bin,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        Split_K); */
    // #pragma omp parallel num_threads(NUM_THREADS)
    // {
    for (int i = 0; i < WARM_UP_ITERATION; i++)
    {
        // #pragma omp parallel num_threads(NUM_THREADS)
        // {
        w2a3_attv_pack_linear_kernel(  0,
                        A, // A_Scale, B_Scale,
                        B,
                        D_bin,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        Split_K, 4);
        // }
    }
    // }
    hipEventRecord(start);
    // #pragma omp parallel num_threads(NUM_THREADS)
    // {
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
    {   
        // #pragma omp parallel num_threads(NUM_THREADS)
        // {
        w2a3_attv_pack_linear_kernel(  0,
                        A, // A_Scale, B_Scale,
                        B,
                        D_bin,
                        M_GLOBAL, N_GLOBAL, K_GLOBAL,
                        Reduction_Workspace,  
                        Split_K, 4);
        // }
    }
    // }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    checkLastCudaError(__LINE__);
    //
    float milliseconds_bin = 0.0f;
    hipEventElapsedTime(&milliseconds_bin, start, stop);
    milliseconds_bin = milliseconds_bin / BENCHMARK_ITERATION;
    float tflops_bin = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_bin / 1000.)) / 1e12;
    half* D_bin_h = NULL;  // col major
    D_bin_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipMemcpy(D_bin_h, D_bin, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_bin);
    hipFree(Reduction_Workspace);
    /////////////////////////////////////////////////////////////////////////////////////////////////
    
    
    // #ifdef BENCHMARK_MODE
    // /////////////////////////////////////////////////////////////////////////////////////////////////
    // printf("Launching FP6-LLM...\n");
    // half* D_fp6 = NULL;
    // hipMalloc(reinterpret_cast<void**>(&D_fp6), sizeof(half) * M_GLOBAL * N_GLOBAL); CheckMallocCUDA(D_fp6);
    // hipMemset(D_fp6, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    // //
    // for (int i = 0; i < WARM_UP_ITERATION; i++)
    //     fp6_linear_kernel(  0,
    //                     (uint4*)A_1bit, A_Scale,
    //                     B_16bit,
    //                     D_fp6,
    //                     M_GLOBAL, N_GLOBAL, K_GLOBAL,
    //                     Reduction_Workspace,  
    //                     Split_K);
    // hipEventRecord(start);
    // for (int i = 0; i < BENCHMARK_ITERATION; i++)
    //     fp6_linear_kernel(  0,
    //                     (uint4*)A_1bit, A_Scale,
    //                     B_16bit,
    //                     D_fp6,
    //                     M_GLOBAL, N_GLOBAL, K_GLOBAL,
    //                     Reduction_Workspace,  
    //                     Split_K);
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // checkLastCudaError(__LINE__);
    // //
    // float milliseconds_fp6 = 0.0f;
    // hipEventElapsedTime(&milliseconds_fp6, start, stop);
    // milliseconds_fp6 = milliseconds_fp6 / BENCHMARK_ITERATION;
    // float tflops_fp6 = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_fp6 / 1000.)) / 1e12;
    // half* D_fp6_h = NULL;  // col major
    // D_fp6_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    // hipMemcpy(D_fp6_h, D_fp6, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    // hipFree(D_fp6);
    // hipFree(Reduction_Workspace);
    // /////////////////////////////////////////////////////////////////////////////////////////////////
    // printf("Verifying correctness of the computations...\n");
    // double totalRelativeError_fp6  = ComputeTotalError(D_cublas_h, D_fp6_h, M_GLOBAL, N_GLOBAL);
    // #endif

    printf("M: %d N: %d K: %d SplitK: %d Iter: %d\n", M_GLOBAL, N_GLOBAL, K_GLOBAL, SPLIT_K, BENCHMARK_ITERATION);
    printf("******************************************Performance*******************************************\n");
    double totalRelativeError_bin = INFINITY;
    #ifdef BENCHMARK_MODE
    printf("Verifying correctness of the computations...\n");
    totalRelativeError_bin  = ComputeTotalError(D_cublas_h, D_bin_h, M_GLOBAL, N_GLOBAL);
    PrintPerformance("cuBLAS", milliseconds_cublas, tflops_cublas, 0.0);
    PrintPerformance("D_fp_h", milliseconds_fp, tflops_fp, INFINITY);
    #endif
    PrintPerformance("BGEMM", milliseconds_bin, tflops_bin, totalRelativeError_bin);
    #ifdef SAVE_IO
        PrintResult("BGEMM", 100, D_cublas_h, D_bin_h, M_GLOBAL, N_GLOBAL);
    #endif
    #ifdef DEBUG_MODE
        PrintMismatch("BGEMM", 100, 0.000, D_cublas_h, D_bin_h, M_GLOBAL, N_GLOBAL);
    #endif

    free(D_cublas_h);
    free(D_bin_h);
    return 0;
}